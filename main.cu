#include "stdlib.h"
#include "hip/hip_runtime.h"

#include "math.h"
#include "stdio.h"

#define MAX_THREADS (512)
#define PI (3.14159265)

__device__  float kernel_sinm(float fn, float fce, float t)
{
	return sin((fn - fce)*t);
	
} 

__device__  float kernel_sinp(float fn, float fce, float t)
{
	return sin((fn + fce)*t);
}

__device__  float kernel_cosm(float fn, float fce, float t)
{
	return cos((fn - fce)*t);
	
} 

__device__  float kernel_cosp(float fn, float fce, float t)
{
	return cos((fn + fce)*t);
}

__global__ void global_calculate_vt(float* fn, float* fce, float* vnp, float* vnm, size_t nh, float* tn, float* vt)
{      
	size_t nt = blockDim.x*blockIdx.x + threadIdx.x;
	float t = tn[nt];
	size_t i;
	float kfce = *fce;
	float sinp;
	float sinm;
	
	//load vn coeff to shared memory
	float vnt = 0;
	for ( i = 0; i < nh; ++i )
	{
		float fnt = fn[i];
		float vnpt = vnp[i];
		float vnmt = vnm[i];
		sinp = kernel_sinp(fnt, kfce, t);
		sinm = kernel_sinm(fnt, kfce, t);
		vnt  += vnpt*sinp + vnmt*sinm;
	}
	vt[nt] = vnt;
}

__global__ void global_calculate_et(float* fn, float* fce, float* enp, float* enm, size_t nh, float* tn, float* et)
{      
	size_t nt = blockDim.x*blockIdx.x + threadIdx.x;
	float t = tn[nt];
	size_t i;
	float kfce = *fce;
	float cosp;
	float cosm;
	
	//load vn coeff to shared memory
	float ent = 0;
	for ( i = 0; i < nh; ++i )
	{
		float fnt = fn[i];
		float enpt = enp[i];
		float enmt = enm[i];
		cosp = kernel_cosp(fnt, kfce, t);
		cosm = kernel_cosm(fnt, kfce, t);
		ent  += enpt*cosp + enmt*cosm;
	}
	et[nt] = ent;
}

void generate_an(float a0, size_t nh, float* an)
{
	size_t i;
	float a = a0*4/PI;
	for (i = 0; i < nh; ++i)
	{
		an[i] = a/(2*i+1);
	}
}

void generate_fn(float fm, size_t nh, float* fn)
{
	size_t i;
	for (i = 0; i < nh; ++i)
	{
		fn[i] = fm*(2*i+1);
	}
}

void generate_vn(float* an, float* fn, float fce,
		 size_t nh, float* vnp, float* vnm)
{
	size_t i;
	float fce2 = fce*fce;
	for (i = 0; i < nh; ++i)
	{
		float fs = fn[i] + fce;
		float fd = fn[i] - fce;
     
		vnp[i] = 0.5*an[i]*(fs/(fce2 - pow(fs,2)));
		vnm[i] = -0.5*an[i]*(fd/(fce2 - pow(fd,2)));
	}	
}

void generate_en(float* an, float* fn, float fce,
		 size_t nh, float* enp, float* enm)
{
	size_t i;
	for (i = 0; i < nh; ++i)
	{     
		enm[i] = 0.5*an[i];
		enp[i] = -0.5*an[i];
	}	
}

void generate_tn(float tstart, float tstop, size_t ntpoints, float* tn)
{
	size_t i;
	float dt = (tstop - tstart)/ntpoints;
	for (i = 0; i < ntpoints; ++i)
	{
		tn[i] = tstart + dt*i;
	}
}



int main(int argc, char** argv)
{
	float fce = 2*PI*atof(argv[1]);
	float fm  = 2*PI*atof(argv[2]);
	float a0  = atof(argv[3]);
	size_t nh  = atoi(argv[4]);
	float tstart = atof(argv[5]);
	float tstop = atof(argv[6]);
	size_t ntpoints = atoi(argv[7]);
	printf("Start the programm!\n");
	printf("Parameters:\n");
	printf("fce = %f Hz,\nfm = %f Hz,\na0 = %f V/cm,\nnh = %d,\ntstart = %f sec.,\ntstop = %f sec.,\nntpoints = %d\n", fce/2/PI, fm/2/PI, a0, (int)nh, tstart, tstop, (int)ntpoints);
	
  
	float* an = (float*)malloc(sizeof(float)*nh);
	float* fn = (float*)malloc(sizeof(float)*nh);
	float* vnp = (float*)malloc(sizeof(float)*nh);
	float* vnm = (float*)malloc(sizeof(float)*nh);
	float* enm = (float*)malloc(sizeof(float)*nh);
	float* enp = (float*)malloc(sizeof(float)*nh);
	float* tn =  (float*)malloc(sizeof(float)*ntpoints);
	float* vt = (float*)malloc(sizeof(float)*ntpoints);
	float* et = (float*)malloc(sizeof(float)*ntpoints);
		
	float* d_vnm = NULL;
	float* d_vnp = NULL;
	float* d_enm = NULL;
	float* d_enp = NULL;
	float* d_tn = NULL;
	float* d_fn = NULL;
	float* d_fce = NULL;
	float* d_sinp = NULL;
	float* d_sinm = NULL;
	float* d_vt = NULL;
	float* d_et = NULL;
	
	size_t i = 0;
	FILE* to;
	
	hipMalloc(&d_fce, sizeof(float));
	hipMalloc(&d_vnm, sizeof(float)*nh);
	hipMalloc(&d_vnp, sizeof(float)*nh);
	hipMalloc(&d_sinp, sizeof(float)*nh);
	hipMalloc(&d_sinm, sizeof(float)*nh);
	hipMalloc(&d_fn, sizeof(float)*nh);
	hipMalloc(&d_tn, sizeof(float)*ntpoints);
	hipMalloc(&d_vt, sizeof(float)*ntpoints);
	hipMalloc(&d_et, sizeof(float)*ntpoints);
	hipMalloc(&d_enm, sizeof(float)*nh);
	hipMalloc(&d_enp, sizeof(float)*nh);

	generate_fn(fm, nh, fn);
	generate_an(a0, nh, an);
	generate_vn(an, fn, fce, nh, vnp, vnm);
	generate_en(an, fn, fce, nh, enp, enm);
	generate_tn(tstart, tstop, ntpoints, tn);

	hipMemcpy(d_vnm, vnm, sizeof(float)*nh, hipMemcpyHostToDevice);
	hipMemcpy(d_vnp, vnp, sizeof(float)*nh, hipMemcpyHostToDevice);
	hipMemcpy(d_enm, enm, sizeof(float)*nh, hipMemcpyHostToDevice);
	hipMemcpy(d_enp, enp, sizeof(float)*nh, hipMemcpyHostToDevice);
	hipMemcpy(d_fn, fn, sizeof(float)*nh, hipMemcpyHostToDevice);
	hipMemcpy(d_fce, &fce, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_tn, tn, sizeof(float)*ntpoints, hipMemcpyHostToDevice);

	global_calculate_vt<<<ntpoints/MAX_THREADS, MAX_THREADS>>>(d_fn, d_fce, d_vnp, d_vnm, nh, d_tn, d_vt);
	global_calculate_et<<<ntpoints/MAX_THREADS, MAX_THREADS>>>(d_fn, d_fce, d_enp, d_enm, nh, d_tn, d_et);
	
	hipMemcpy(vt, d_vt, sizeof(float)*ntpoints, hipMemcpyDeviceToHost);
	hipMemcpy(et, d_et, sizeof(float)*ntpoints, hipMemcpyDeviceToHost);
	
	to = fopen("vt.dat", "w");
	for(i = 0; i < ntpoints; ++i)
	{
		fprintf(to, "%e\t%e\n", tn[i], vt[i]);
	}
	fclose(to);

	to = fopen("et.dat", "w");
	for(i = 0; i < ntpoints; ++i)
	{
		fprintf(to, "%e\t%e\n", tn[i], et[i] + sin(fce*tn[i]));
	}
	fclose(to);
	

	free(vnp);
	free(vnm);
	free(an);
	free(fn);
	free(tn);
	free(vt);
	free(et);
	free(enm);
	free(enp);
	hipFree(d_vnm);
	hipFree(d_vnp);
	hipFree(d_enm);
	hipFree(d_enp);
	hipFree(d_sinm);
	hipFree(d_sinp);
	hipFree(d_fn);
	hipFree(d_tn);
	hipFree(d_fce);
	hipFree(d_vt);

	printf("The programm is done!\n");
}
